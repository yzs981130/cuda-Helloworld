#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void hello()
{
	printf("Hello from block %d\n", blockIdx.x);
}
int main()
{
	hello <<<16, 1>>> ();
	hipDeviceSynchronize();
	printf("Bye!\n");
	return 0;
}